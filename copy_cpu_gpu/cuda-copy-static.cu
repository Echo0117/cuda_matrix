#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

#define N 1024

// Define an static array dA[N] of floats on the GPU
// Definir un tableau de float dA[N] de taille statique sur le GPU
// TODO / A FAIRE ...
float A[N];
__device__ float dA[N];


int main() {
  float A[N], B[N];
  int i;

  // Initialization
  // Initialisation
  for (i = 0; i < N; i++) { A[i] = (float)i; }

  // cudaMemcpy from A[N] to dA[N]
  // cudaMemcpy de A[N] vers dA[N]
  // TODO / A FAIRE ...
  hipMemcpyToSymbol(HIP_SYMBOL(dA), A, sizeof(float) * N, 0, hipMemcpyHostToDevice);
  // cudaMemcpy from dA[N} to B[N]
  // cudaMemcpy de dA[N] vers B[N]
  // TODO / A FAIRE ...
  hipMemcpyFromSymbol(B, HIP_SYMBOL(dA), sizeof(float) * N, 0, hipMemcpyDeviceToHost);

  for (i = 0; i < N; i++) { cout<<B[i];}
  // Wait for GPU kernels to terminate
  // Attendre que les kernels GPUs terminent
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verify the results
  // Verifier le resultat
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "The copy is incorrect!\n"; }
  else { cout << "The copy is correct!\n"; }

  return 0;
}
